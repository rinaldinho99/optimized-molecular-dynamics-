
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <cstdlib>
#include <time.h>

#define THREADS 64
#define SIZE 1024
#define BLOCK_SIZE 512
#define TILE_SIZE (BLOCK_SIZE * 2)
// El numero de particulas a reducir podra ser hasta 2 * SIZE
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
__device__ double dist(int nd, double r1, double r2, double dr[]);
void r8mat_uniform_ab(int m, int n, double a, double b, int *seed, double r[]);
__global__ void update(int np, int nd, double pos[], double vel[], double f[], double acc[], double mass, double dt);
__global__ void compute(int np, int nd, double pos[], double vel[], double mass, double f[], double pot[], double kin[]);
__global__ void reduction(double* g_data, int n, double* out);

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[]) {
    double *d_acc = NULL;
    double dt;
    double e0;
    double *d_force = NULL;
    double mass = 1.0;
    int nd;
    int np;
    double *d_pos = NULL;
    int step_num;
    int step_print = 0;
    int step_print_index = 0;
    int step_print_num = 10;
    double *d_vel = NULL;



    struct timeval tv_start, tv_end;
    double run_time;

    printf("\nMD\n");
    printf("  CUDA version\n");
    printf("  A molecular dynamics program.\n");

    if (argc > 1)
        nd = atoi(argv[1]);
    else {
        printf("\nEnter ND, the spatial dimension (2 or 3).\n");
        scanf("%d", &nd);
    }

    if (argc > 2)
        np = atoi(argv[2]);
    else {
        printf("\nEnter NP, the number of particles (500, for instance).\n");
        scanf("%d", &np);
    }

    if (argc > 3)
        step_num = atoi(argv[3]);
    else {
        printf("\nEnter ND, the number of time steps (500 or 1000, for instance).\n");
        scanf("%d", &step_num);
    }

    if (argc > 4)
        dt = atof(argv[4]);
    else {
        printf("\nEnter DT, the size of the time step (0.1, for instance).\n");
        scanf("%lf", &dt);
    }

    printf("\nND, the spatial dimension, is %d\n", nd);
    printf("NP, the number of particles in the simulation, is %d\n", np);
    printf("STEP_NUM, the number of time steps, is %d\n", step_num);
    printf("DT, the size of each time step, is %lf\n", dt);

   double* h_acc;
   double* h_force;
   double* h_pos;
   double* h_vel;
   double* h_ken;
   double* h_pen;
   double* d_ken = NULL;
   double* d_pen = NULL;
   double ken_res;
   double pen_res;

   unsigned int size = nd * np * sizeof(double);
   unsigned int size_part = np * sizeof(double);

    h_acc = (double *)malloc(size);
    h_force = (double *)malloc(size);
    h_pos = (double *)malloc(size);
    h_vel = (double *)malloc(size);
    h_ken = (double *)malloc(size_part);
    h_pen = (double *)malloc(size_part);

    hipMalloc(&d_acc, nd * np * sizeof(double));
    hipMalloc(&d_force, nd * np * sizeof(double));
    hipMalloc(&d_pos, nd * np * sizeof(double));
    hipMalloc(&d_vel, nd * np * sizeof(double));

    printf("\nAt each step, we report the potential and kinetic energies.\n");
    printf("The sum of these energies should be a constant.\n");
    printf("As an accuracy check, we also print the relative error\n");
    printf("in the total energy.\n");
    printf("\n      Step      Potential       Kinetic        (P+K-E0)/E0\n");
    printf("                Energy P        Energy K       Relative Energy Error\n");
    printf("\n");

    step_print = 0;
    step_print_index = 0;
    step_print_num = 10;

    gettimeofday(&tv_start, NULL);

    int j;
    int seed = 123456789;
    r8mat_uniform_ab(nd, np, 0.0, 10.0, &seed, h_pos);

    for (int j = 0; j < np; j++) {
        for (int i = 0; i < nd; i++) {
            h_vel[i + j * nd] = 0.0;
            h_acc[i + j * nd] = 0.0;
        }
    }

    for(j = 0 ; j < np ; j++){
        h_ken[j] = 0;
        h_pen[j] = 0;
    }

    hipMalloc(&d_acc, size);
    hipMalloc(&d_force, size);
    hipMalloc(&d_pos, size);
    hipMalloc(&d_vel, size);
    hipMalloc(&d_ken, size_part);
    hipMalloc(&d_pen, size_part);

    hipMemcpy(d_acc, h_acc, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pos, h_pos, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vel, h_vel, size, hipMemcpyHostToDevice);
    hipMemset(d_force, 0, size);
    hipMemset(d_ken, 0, size_part);
    hipMemset(d_pen, 0, size_part);


    int bDim = THREADS;   // numThreads = 64
    int gDim = np / bDim; // numBlocks
    int shared_memory_size = THREADS * nd * sizeof(double);


    for (int step = 0; step <= step_num; step++) {
        if (step > 0) {
            update<<<gDim, THREADS>>>(np, nd, d_pos, d_vel, d_force, d_acc, mass, dt);
        }
        compute<<<gDim, bDim, shared_memory_size>>>(np, nd, d_pos, d_vel, mass, d_force, d_pen, d_ken);



        if (step == step_num / 10 * (step / (step_num / 10))){
            ken_res = 0.0;
            pen_res = 0.0;

            double* d_pen_res;
            double* d_ken_res;
            hipMalloc((void**)&d_pen_res, sizeof(double));
            hipMalloc((void**)&d_ken_res, sizeof(double));
        
            hipMemcpy(d_pen_res, &pen_res, sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_ken_res, &ken_res, sizeof(double), hipMemcpyHostToDevice);
        
            int blocks = ceil(np / (double)TILE_SIZE);

            reduction <<<blocks, BLOCK_SIZE>>>(d_pen, np, d_pen_res);
            reduction <<<blocks, BLOCK_SIZE>>>(d_ken, np, d_ken_res);

            hipMemcpy(&pen_res, d_pen_res, sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(&ken_res, d_ken_res, sizeof(double), hipMemcpyDeviceToHost);
            hipFree(d_pen_res);
            hipFree(d_ken_res);
            
            if (step == 0) {
                e0 = ken_res + pen_res;
            }


            printf("  %8d  %14f  %14f  %14e\n", step, pen_res, ken_res, (pen_res + ken_res - e0) / e0);
            step_print += 2;
            step_print_index = step_print_index + 1;
            step_print = (step_print_index * step_num) / step_print_num;
        }
    }


    gettimeofday(&tv_end, NULL);
    run_time = (tv_end.tv_sec - tv_start.tv_sec) * 1000000 +
               (tv_end.tv_usec - tv_start.tv_usec); // en us
    run_time = run_time / 1000000;                  // en s

    printf("\n Tiempo version CUDA = %lg s\n", run_time);

    free(h_acc);
    free(h_force);
    free(h_pos);
    free(h_vel);
    free(h_pen);   
    free(h_ken);

    hipFree(d_acc);
    hipFree(d_force);
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(d_ken);
    hipFree(d_pen);

    printf("\nMD\n");
    printf("  Normal end of execution.\n");
    printf("\n");

    return 0;
}
__device__ double dist(int nd, double *r1, double *r2, double *dr) {
    double d = 0.0;
    for (int i = 0; i < nd; i++) {
        dr[i] = r1[i] - r2[i];
        d += dr[i] * dr[i];
    }
    return sqrt(d);
}

void r8mat_uniform_ab(int m, int n, double a, double b, int *seed, double r[]) {
    const int i4_huge = 2147483647;

    if (*seed == 0) {
        fprintf(stderr, "\nR8MAT_UNIFORM_AB - Fatal error!\n");
        fprintf(stderr, "  Input value of SEED = 0.\n");
        exit(1);
    }

    for (int j = 0; j < n; j++) {
        for (int i = 0; i < m; i++) {
            int k = *seed / 127773;
            *seed = 16807 * (*seed - k * 127773) - k * 2836;

            if (*seed < 0) {
                *seed += i4_huge;
            }

            r[i + j * m] = a + (b - a) * (double)(*seed) * 4.656612875E-10;
        }
    }
} 

__global__ void update(int np, int nd, double *pos, double *vel, double *f, double *acc, double mass, double dt) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= np) return;

    double rmass = 1.0 / mass;
    for (int i = 0; i < nd; i++) {
        pos[i + k * nd] = pos[i + k * nd] + vel[i + k * nd] * dt + 0.5 * acc[i + k * nd] * dt * dt;
        vel[i + k * nd] = vel[i + k * nd] + 0.5 * dt * (f[i + k * nd] * rmass + acc[i + k * nd]);
        acc[i + k * nd] = f[i + k * nd] * rmass;
    }
}

__global__ void compute(int np, int nd, double pos[], double vel[], double mass, double f[], double pot[], double kin[]) {
    extern __shared__ double shared_pos[];

    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    if (k >= np) return;

    double ke = 0.0;
    double pe = 0.0;
    double PI2 = 1.570796327; // 3.141592653589793 / 2.0
    double rij[3];

    // Load particle positions into shared memory
    for (int i = 0; i < nd; i++) {
        shared_pos[tid * nd + i] = pos[k * nd + i];
    }
    __syncthreads();

    // Initialize force to zero
    for (int i = 0; i < nd; i++) {
        f[i + k * nd] = 0.0;
    }

    // Compute forces and potential energy
    for (int j = 0; j < np; j++) {
        if (k != j) {
            double *pos_k = &shared_pos[tid * nd];
            double *pos_j = &pos[j * nd];

            double d_squared = 0.0;
            for (int i = 0; i < nd; i++) {
                rij[i] = pos_k[i] - pos_j[i];
                d_squared += rij[i] * rij[i];
            }

            if (d_squared > 0.0) {
                double d = sqrt(d_squared);
                double d2 = (d < PI2) ? d : PI2;
                double sin_d2 = sin(d2);
                double sin_2d2 = sin(2.0 * d2);

                pe += 0.5 * sin_d2 * sin_d2;

                for (int i = 0; i < nd; i++) {
                    f[i + k * nd] -= rij[i] * sin_2d2 / d;
                }
            }
        }
    }

    // Compute kinetic energy
    for (int i = 0; i < nd; i++) {
        ke += vel[k * nd + i] * vel[k * nd + i];
    }
    ke *= 0.5 * mass;

    // Store potential and kinetic energies
    pot[k] = pe;
    kin[k] = ke;
}


__global__ void reduction(double *g_data, int n, double* out) {
    int tile = TILE_SIZE * blockIdx.x;
    __shared__ double data[BLOCK_SIZE];

    int my_idx = tile + threadIdx.x;
    data[threadIdx.x] = (my_idx < n) ? g_data[my_idx] : 0.0;

    int next_idx = my_idx + blockDim.x;
    if (next_idx < n) {
        data[threadIdx.x] += g_data[next_idx];
    }

    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(out, data[0]);
    }
}
